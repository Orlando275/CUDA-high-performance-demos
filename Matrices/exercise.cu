
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N  64

__global__ void matrixMulGPU( int * a, int * b, int * c )
{
    int col=blockDim.x*blockIdx.x+threadIdx.x;
    int row=blockDim.y*blockIdx.y+threadIdx.y;
    if (row<N&&col<N)
    {
        int val=0;
        for (int k = 0; k < N; k++)
        {
            val+=a[row*N+k]*b[k*N+col];
        }
        c[row*N+col]=val;
    }
    

}

void matrixMulCPU( int * a, int * b, int * c )
{
  int val = 0;

  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      val = 0;
      for ( int k = 0; k < N; ++k )
        val += a[row * N + k] * b[k * N + col];
      c[row * N + col] = val;
    }
}

int main()
{
  int *a, *b, *c_cpu, *c_gpu;
  int size = N * N * sizeof (int); 

  hipMallocManaged (&a, size);
  hipMallocManaged (&b, size);
  hipMallocManaged (&c_cpu, size);
  hipMallocManaged (&c_gpu, size);

  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      a[row*N + col] = row;
      b[row*N + col] = col+2;
      c_cpu[row*N + col] = 0;
      c_gpu[row*N + col] = 0;
    }
    

  dim3 threads_per_block(8,8);
  dim3 number_of_blocks((N+threads_per_block.x-1)/threads_per_block.x,(N+threads_per_block.y-1)/threads_per_block.y);

  matrixMulGPU <<< number_of_blocks, threads_per_block >>> ( a, b, c_gpu );

  hipDeviceSynchronize();

  matrixMulCPU( a, b, c_cpu );

  bool error = false;
  for( int row = 0; row < N && !error; ++row )
    for( int col = 0; col < N && !error; ++col )
      if (c_cpu[row * N + col] != c_gpu[row * N + col])
      {
        printf("FOUND ERROR at c[%d][%d]\n", row, col);
        error = true;
        break;
      }
  if (!error)
    printf("Success!\n");

  hipFree(a); hipFree(b);
  hipFree( c_cpu ); hipFree( c_gpu );
}
