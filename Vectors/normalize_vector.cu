#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cmath>
#define CHECK_CUDA(call)                                                 \
    {                                                                    \
        hipError_t err = (call);                                        \
        if (err != hipSuccess)                                          \
        {                                                                \
            fprintf(stderr, "CUDA error at %s:%d in %s: %s\n",           \
                    __FILE__, __LINE__, #call, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                          \
        }                                                                \
    }
__global__ void normalize_Vector_Sharememory(float *Vector, float *sumT, int N)
{
    extern __shared__ float cache[];
    int tid = threadIdx.x;

    int idx = blockDim.x * blockIdx.x + tid;
    cache[tid] = (idx < N) ? Vector[idx] * Vector[idx] : 0.0f;
    __syncthreads();

    for (int i = blockDim.x / 2; i > 32; i >>= 1)
    {
        if (tid < i)
        {
            cache[tid] += cache[tid + i];
        }
        __syncthreads();
    }

    float val = cache[tid];
    if (tid < 32)
    {
        for (int i = 16; i > 0; i >>= 1)
        {
            val += __shfl_down_sync(0xffffffff, val, i);
        }
    }
    if (tid == 0)
    {
        sumT[blockIdx.x] = val;
    }
}

__global__ void result_Total_Parcial_Sum(float *parcialSum, float *sumTotal, int N)
{
    extern __shared__ float cache[];
    int tid = threadIdx.x;
    int idx = blockDim.x * blockIdx.x + tid;
    cache[tid] = (idx < N) ? parcialSum[idx] : 0.0f;

    __syncthreads();

    for (int i = blockDim.x / 2; i > 32; i >>= 1)
    {
        if (tid < i)
        {
            cache[tid] += cache[tid + i];
        }
        __syncthreads();
    }
    float val = cache[tid];
    if (tid < 32)
    {
        for (int i = 16; i > 0; i >>= 1)
        {
            val += __shfl_down_sync(0xffffffff, val, i);
        }
    }
    if (tid == 0)
    {
        sumTotal[0] = val;
    }
}

__global__ void aplly_Normalize(float *vectorResult, float *vecInit, float raizX, int N)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N)
    {
        vectorResult[idx] = vecInit[idx] / raizX;
    }
}

int main()
{
    int N = 2 << 24;
    float sumCuadrados = 0;

    float *arrA = new float[N];
    float *arrResult = new float[N];
    for (int i = 0; i < N; i++)
    {
        *(arrA + i) = i;
    }
    float *D_A;
    float *D_S;
    float *D_Result;

    int blockSize = 32*32;
    int numBlock = (N + blockSize - 1) / blockSize;

    CHECK_CUDA(hipMalloc((void **)&D_A, N * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **)&D_S, numBlock * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **)&D_Result, N * sizeof(float)));

    CHECK_CUDA(hipMemcpy(D_A, arrA, N * sizeof(float), hipMemcpyHostToDevice));

    normalize_Vector_Sharememory<<<numBlock, blockSize, blockSize * sizeof(float)>>>(D_A, D_S, N);
    CHECK_CUDA(hipDeviceSynchronize());


    int threadsFinal = min(256, numBlock);

    result_Total_Parcial_Sum<<<1, threadsFinal, threadsFinal * sizeof(float)>>>(D_S, D_Result, numBlock);
    CHECK_CUDA(hipDeviceSynchronize());


    CHECK_CUDA(hipMemcpy(&sumCuadrados, D_Result, sizeof(float), hipMemcpyDeviceToHost));
    float raizX = sqrt((sumCuadrados));

    aplly_Normalize<<<numBlock, blockSize>>>(D_Result, D_A, raizX, N);
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(arrResult, D_Result, N*sizeof(float), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(D_A));
    CHECK_CUDA(hipFree(D_S));
    CHECK_CUDA(hipFree(D_Result));
    delete[] arrA;
    delete[] arrResult;
    return 0;
}
