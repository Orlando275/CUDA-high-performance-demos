
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
using namespace std;
#define CHECK_CUDA(call) { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d in %s: %s\n", \
                __FILE__, __LINE__, #call, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

__global__ void sumVect(float* A,float* B,float* C,int N){
int idx=blockIdx.x*blockDim.x+threadIdx.x;
int stride = blockDim.x * gridDim.x;

for (int i = idx; i < N; i += stride) {
        C[i] = A[i] + B[i];
    }

}

int main(){
    int n = 2<<24;
    float* arrA=new float[n];
    float* arrB=new float[n];
    float* arrC=new float[n];

    for (int i = 0; i < n; i++)
    {
        *(arrA+i)=i+2;
        *(arrB+i)=i+2;
    }
    float *d_A,*d_B,*d_C;
    CHECK_CUDA(hipMalloc((void**)&d_A,n*sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_B,n*sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_C,n*sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_B, arrB, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_A, arrA, n * sizeof(float), hipMemcpyHostToDevice));


    int deviceId;
    hipGetDevice(&deviceId);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props,deviceId);

    int warpSize= props.warpSize;

    int numBlock=32*32;
    int blockSize=warpSize;
    
    sumVect<<<numBlock,blockSize>>>(d_A,d_B,d_C,n);
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipMemcpy(arrC,d_C,n*sizeof(float),hipMemcpyDeviceToHost));
    

    delete[]arrB;
    delete[]arrA;
    delete[]arrC;
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    return 0;
}